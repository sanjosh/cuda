
#include <hip/hip_runtime.h>
#include <stdio.h>

__host__ __device__ void sq(int *a, int numElements)
{
#ifdef __CUDA_ARCH__
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	a[idx] = a[idx] * a[idx];
#else
	printf("host side\n");
#endif	
}

__global__ void callme(int *a, int numElements)
{
	sq(a, numElements);	
}

int main()
{
	int *a;
	int numElements = 1024;
	hipMalloc((void**)a, numElements);
	int blockSize = 16;
	int numBlocks = numElements/blockSize;
	callme <<<numBlocks, blockSize>>> (a, numElements);
	if (hipDeviceSynchronize() != hipSuccess) {
		printf("error\n");
	}
	hipFree(a);
}
