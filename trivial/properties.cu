#include <hip/hip_runtime.h>

int main() {

	hipInit(0);  

	int cnt;
	hipGetDeviceCount( &cnt );

	for (int it = 0; it < cnt ; ++ it) {

		hipDevice_t dev;
		char name[1024];
		int major, minor;
		size_t bytes;
		CUdevprop properties;

		hipDeviceGet              ( &dev, it );
		hipDeviceGetName          ( name, sizeof(name), dev );
		hipDeviceComputeCapability( &major, &minor,     dev );
		hipDeviceTotalMem         ( &bytes,             dev );
		cuDeviceGetProperties    ( &properties,        dev );
		// or cuDeviceGetAttribute
	}
}
